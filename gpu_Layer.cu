#include "Layer.h"

Layer::Layer(const std::string &_network, const std::string &_name, const std::string &_type, bool _ReLU, int _stride,
        int _padding) : ReLU(_ReLU), stride(_stride), padding(_padding) {
    this->network = _network;
    this->name = _name;
    this->type = _type;
	this->init = false;
}

Layer::~Layer() {
	if(init) {
		hipHostFree(weights);
		hipHostFree(bias);
		hipHostFree(activations);
		hipHostFree(output_activations);
	}
}

float Layer::act_get(int i, int j, int k, int l) const {
    uint32_t index = act_shape[1]*act_shape[2]*act_shape[3]*i + act_shape[2]*act_shape[3]*j + act_shape[3]*k + l;
    return activations[index];
}

float Layer::wgt_get(int i, int j, int k, int l) const {
    uint32_t index = wgt_shape[1]*wgt_shape[2]*wgt_shape[3]*i + wgt_shape[2]*wgt_shape[3]*j + wgt_shape[3]*k + l;
    return weights[index];
}

uint64_t getMaxIndex(const std::string &array) const {
    if(array == "weights") {
        return wgt_shape[0]*wgt_shape[1]*wgt_shape[2]*wgt_shape[3];
    } else if(array == "bias") {
        return bias_shape[0];
    } else if(array == "activations") {
        #ifdef FORCE_ONE_IMAGE
        return 1*act_shape[1]*act_shape[2]*act_shape[3];
        #else
        return act_shape[0]*act_shape[1]*act_shape[2]*act_shape[3];
        #endif
    } else if(array == "output_activations") {
        #ifdef FORCE_ONE_IMAGE
        if(out_act_shape.size() == 4) return 1*out_act_shape[1]*out_act_shape[2]*out_act_shape[3];
        else return 1*out_act_shape[1];
        #else
        if(out_act_shape.size() == 4) return out_act_shape[0]*out_act_shape[1]*out_act_shape[2]*out_act_shape[3];
        else return out_act_shape[0]*out_act_shape[1];
        #endif
    } else return 0;
}

void Layer::zero_pad() {

    #ifdef FORCE_ONE_IMAGE
    auto batch_size = (unsigned)1;
    #else
    auto batch_size = act_shape[0];
    #endif
    int act_channels = act_shape[1];
    int Nx = act_shape[2];
    int Ny = act_shape[3];
    int new_Nx = Nx + 2*padding;
    int new_Ny = Ny + 2*padding;

    uint64_t new_max_index = batch_size * act_channels * new_Nx * new_Ny;
    float* tmp_activations;
    hipHostMalloc((void **) &tmp_activations, new_max_index * sizeof(float));
    if (tmp_activations == NULL) {
        fprintf(stderr, "Error: Failed to allocate padded activations!\n");
        exit(EXIT_FAILURE);
    }

    for(uint64_t i = 0; i < new_max_index; i++) {
        tmp_activations[i] = 0;
    }

    for(int n = 0; n < batch_size; n++) {
        for (int k = 0; k < act_channels; k++) {
            for (int i = 0; i < Nx; i++) {
                for(int j = 0; j < Ny; j++) {
                    uint32_t index_out = act_channels*new_Nx*new_Ny*n + new_Nx*new_Ny*k + new_Ny*(padding + i) +
                            (padding + j);
                    uint32_t index_in = act_channels*Nx*Ny*n + Nx*Ny*k + Ny*i + j;
                    tmp_activations[index_out] = activations[index_in];
                }
            }
        }
    }

    hipHostFree(activations);
    activations = tmp_activations;
    act_shape.clear();
    act_shape.push_back(batch_size);
    act_shape.push_back(act_channels);
    act_shape.push_back(new_Nx);
    act_shape.push_back(new_Ny);

}

void Layer::act_split_4D(int K, int X, int Y) {

    #ifdef FORCE_ONE_IMAGE
    auto batch_size = (unsigned)1;
    #else
    auto batch_size = act_shape[0];
    #endif
    int act_channels = act_shape[1];
    int Nx = act_shape[2];
    int Ny = act_shape[3];

    uint64_t new_max_index = batch_size * K * X * Y;
    float* tmp_activations;
    hipHostMalloc((void **) &tmp_activations, new_max_index * sizeof(float));
    if (tmp_activations == NULL) {
        fprintf(stderr, "Error: Failed to allocate padded activations!\n");
        exit(EXIT_FAILURE);
    }

    for(int n = 0; n < batch_size; n++) {
        for (int k = 0; k < act_channels; k++) {
            for (int i = 0; i < Nx; i++) {
                for(int j = 0; j < Ny; j++) {
                    int new_k = k / (X*Y);
                    int rem = k % (X*Y);
                    int new_i = rem / Y;
                    int new_j = rem % Y;
                    uint32_t index_out = K*X*Y*n + X*Y*new_k + Y*new_i + new_j;
                    uint32_t index_in = act_channels*Nx*Ny*n + Nx*Ny*k + Ny*i + j;
                    tmp_activations[index_out] = activations[index_in];
                }
            }
        }
    }

    hipHostFree(activations);
    activations = tmp_activations;
    act_shape.clear();
    act_shape.push_back(batch_size);
    act_shape.push_back((unsigned)K);
    act_shape.push_back((unsigned)X);
    act_shape.push_back((unsigned)Y);

}

void Layer::wgt_split_4D(int K, int X, int Y) {

    int num_filters = wgt_shape[0];
    int wgt_channels = wgt_shape[1];
    int Kx = wgt_shape[2];
    int Ky = wgt_shape[3];

    uint64_t new_max_index = num_filters * K * X * Y;
    float* tmp_weights;
    hipHostMalloc((void **) &tmp_weights, new_max_index * sizeof(float));
    if (tmp_weights == NULL) {
        fprintf(stderr, "Error: Failed to allocate padded weights!\n");
        exit(EXIT_FAILURE);
    }

    for(int n = 0; n < num_filters; n++) {
        for (int k = 0; k < wgt_channels; k++) {
            for (int i = 0; i < Kx; i++) {
                for(int j = 0; j < Ky; j++) {
                    int new_k = k / (X*Y);
                    int rem = k % (X*Y);
                    int new_i = rem / Y;
                    int new_j = rem % Y;
                    uint32_t index_out = K*X*Y*n + X*Y*new_k + Y*new_i + new_j;
                    uint32_t index_in = wgt_channels*Kx*Ky*n + Kx*Ky*k + Ky*i + j;
                    tmp_weights[index_out] = weights[index_in];
                }
            }
        }
    }

    hipHostFree(weights);
    weights = tmp_weights;
    wgt_shape.clear();
    wgt_shape.push_back(num_filters);
    wgt_shape.push_back((unsigned)K);
    wgt_shape.push_back((unsigned)X);
    wgt_shape.push_back((unsigned)Y);

}

void Layer::reshape_to_2D() {

    #ifdef FORCE_ONE_IMAGE
    auto batch_size = (unsigned)1;
    #else
    auto batch_size = act_shape[0];
    #endif
    int act_channels = act_shape[1];
    int Nx = act_shape[2];
    int Ny = act_shape[3];
    int new_act_channels = act_channels * Nx * Ny;

    act_shape.clear();
    act_shape.push_back(batch_size);
    act_shape.push_back(new_act_channels);
    act_shape.push_back(1);
    act_shape.push_back(1);

}

inline
hipError_t check_error(hipError_t err, std::string task) {
  if (err != hipSuccess) {
    fprintf(stderr, "Error: Failed to %s (error code: %s)!\n", task.c_str(), hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return err;
}

// Read network from numpy arrays
void Layer::read_layer() {

    cnpy::NpyArray data_npy;
    uint64_t max_index;

    cnpy::npy_load("net_traces/" + network + "/wgt-" + name + ".npy" , data_npy, wgt_shape);
    max_index = getMaxIndex("weights");
    check_error(hipHostMalloc((void **) &weights, max_index * sizeof(float)),"allocate layer weights");

    for(uint32_t i = 0; i < max_index; i++)
        weights[i] = data_npy.data<float>()[i];

    cnpy::npy_load("net_traces/" + network + "/bias-" + name + ".npy" , data_npy, bias_shape);
    max_index = getMaxIndex("bias");
    check_error(hipHostMalloc((void **) &bias, max_index * sizeof(float)),"allocate layer bias");

    for(uint32_t i = 0; i < max_index; i++)
        bias[i] = data_npy.data<float>()[i];

    cnpy::npy_load("net_traces/" + network + "/act-" + name + "-0.npy" , data_npy, act_shape);
    max_index = getMaxIndex("activations");
    check_error(hipHostMalloc((void **) &activations, max_index * sizeof(float)),"allocate layer activations");

    for(uint32_t i = 0; i < max_index; i++)
        activations[i] = data_npy.data<float>()[i];

    cnpy::npy_load("net_traces/" + network + "/act-" + name + "-0-out.npy" , data_npy, out_act_shape);
    max_index = getMaxIndex("output_activations");
    check_error(hipHostMalloc((void **) &output_activations, max_index * sizeof(float)),"allocate layer output activations");

    for(uint32_t i = 0; i < max_index; i++)
        output_activations[i] = data_npy.data<float>()[i];

	this->init = true;
	
	#ifdef VERBOSE
    printf("Layer %s loaded into memory\n",name.c_str());
	#endif

}
