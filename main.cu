#include "hip/hip_runtime.h"

#include "Layer.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

#define GLOBAL_TIME
//#define VERBOSE

struct host_data {
	std::vector<float*> wgt_queue;
	std::vector<int*> wgt_queue_k;
    std::vector<int*> wgt_queue_r;
    std::vector<int*> wgt_queue_s;
    std::vector<int> wgt_queue_size;
};

struct device_data {
	float *act;

	float *act_queue; 
    int *act_queue_x;
	int *act_queue_y;
    
	float *wgt_queue;
    int *wgt_queue_k;
	int *wgt_queue_r;
	int *wgt_queue_s;
};

//############################################### Read networks ########################################################

std::vector<Layer> read_bvlc_alexnet() {
    std::vector<Layer> network;
    network.push_back(Layer("bvlc_alexnet","conv1","conv",true,4,0));
    network.push_back(Layer("bvlc_alexnet","conv2","conv",true,1,2));
    network.push_back(Layer("bvlc_alexnet","conv3","conv",true,1,1));
    network.push_back(Layer("bvlc_alexnet","conv4","conv",true,1,1));
    network.push_back(Layer("bvlc_alexnet","conv5","conv",true,1,1));
    network.push_back(Layer("bvlc_alexnet","fc6","fc",true,1,0));
    network.push_back(Layer("bvlc_alexnet","fc7","fc",true,1,0));
    network.push_back(Layer("bvlc_alexnet","fc8","fc",false,1,0));
    return network;
}

std::vector<Layer> read_vgg_cnn_s() {
    std::vector<Layer> network;
    network.push_back(Layer("vgg_cnn_s","conv1","conv",true,2,0));
    network.push_back(Layer("vgg_cnn_s","conv2","conv",true,1,0));
    network.push_back(Layer("vgg_cnn_s","conv3","conv",true,1,1));
    network.push_back(Layer("vgg_cnn_s","conv4","conv",true,1,1));
    network.push_back(Layer("vgg_cnn_s","conv5","conv",true,1,1));
    network.push_back(Layer("vgg_cnn_s","fc6","fc",true,1,0));
    network.push_back(Layer("vgg_cnn_s","fc7","fc",true,1,0));
    network.push_back(Layer("vgg_cnn_s","fc8","fc",false,1,0));
    return network;
}

//############################################### Auxiliary functions ##################################################
double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

inline
hipError_t check_error(hipError_t err, std::string task) {
  if (err != hipSuccess) {
    fprintf(stderr, "Error: Failed to %s (error code: %s)!\n", task.c_str(), hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return err;
}

void check_grid(dim3 grid, std::string kernel){
    if(grid.x >65535 || grid.y >65535 ||grid.z >65535){
        printf("Kernel:%s...Wrong grid assignment\n",kernel.c_str());
        exit(EXIT_FAILURE);
    }
}

template <typename T>
void print(const T a, int size = 20){
    for(int i=0; i<size;i++)
        printf("%4.4f;",a[i]);
    printf("\n");
}

template <typename T>
T* host2Dev(uint64_t size, const T *h_data, std::string task){
    T* d_data;
    check_error(hipMalloc((void**) &d_data, size*sizeof(T)),task);
    check_error(hipMemcpy(d_data, h_data, size*sizeof(T), hipMemcpyHostToDevice),task);

    return d_data;
}

// Checking function
void check_values(const Layer &layer, const float *output_activations, float min_error = 0.01) {

    #ifdef VERBOSE
    printf("Checking values for layer: %s of type %s\n",layer.name.c_str(),layer.type == "conv" ? "convolution" :
            "fully connected");
    uint32_t count = 0;
    #endif
    for(uint32_t i = 0; i < layer.getMaxIndex("output_activations"); i++) {
        #ifdef VERBOSE
        if(fabsf(output_activations[i] - layer.output_activations[i]) > min_error) count++;
		#else
		assert(fabsf(output_activations[i] - layer.output_activations[i]) <= min_error);
		#endif
    }
	#ifdef VERBOSE
    printf("ERRORS: %u out of %lu with absolute error tolerance of %.2f\n\n",count,
            layer.getMaxIndex("output_activations"), min_error);
	#endif
}

//############################################### CUDA SCNN ############################################################

//naive implementation
__global__ void kAddBias(int n, int K, int W, int H, const float *d_bias, float *d_output_activations){

    int h = threadIdx.x + blockIdx.x*blockDim.x;
    int w = threadIdx.y + blockIdx.y*blockDim.y;
    int k = threadIdx.z + blockIdx.z*blockDim.z;

    //TODO: try different configurations
    if(k < K && w < W && h < H){
        int pos = n*K*W*H + k * W * H + w * H + h;
        d_output_activations[pos] = d_bias[k];
    }
}

__global__ void kRelu(int N, int K, int W, int H, float *d_output_activations){
    
    int x = threadIdx.x + blockIdx.x*blockDim.x;

    if(x < N*K*W*H){
        d_output_activations[x] = fmaxf(d_output_activations[x],0);
    }
}

//naive implementation
__global__ void kPopulate_effectual_activations(int n, int channel, int sx, int sy, int C, int X, int Y, int stride,
        device_data dev, int *act_queue_size) {

    int y = threadIdx.x + blockIdx.x*blockDim.x;
    int x = threadIdx.y + blockIdx.y*blockDim.y;

    if(x < X){
        int tmp_sx = x & (stride-1);
        if(y < Y){
            int pos = C*X*Y*n + X*Y*channel + x*Y + y;
            int tmp_sy = y & (stride-1);
            float act_bits = dev.act[pos];
            if(act_bits !=0 && sx == tmp_sx && sy == tmp_sy){
                int index = atomicAdd(act_queue_size,1);
                dev.act_queue[index] = act_bits;
                dev.act_queue_x[index] = x;
                dev.act_queue_y[index] = y;
            }
        }
    }
}

static __device__ __forceinline__ unsigned int log2(unsigned int a){
    return (a) ? (__float_as_int(__uint2float_rz(a)) >> 23) - 127 : 0;
}

//naive implmentation
__global__ void kComputePE(int n, int W, int H, int K, unsigned int stride, int *act_queue_size, int wgt_queue_size, 
		float *d_wgt_queue, int *d_wgt_queue_k, int *d_wgt_queue_r, int *d_wgt_queue_s, int size_eff, device_data dev, float *d_output_activations) {
    //TODO: use shared mem.
    //TODO: try different configurations
    /*__shared__ float s_wgt_queue[10];
    __shared__ int s_wgt_queue_k[10];
    __shared__ int s_wgt_queue_r[10];
    __shared__ int s_wgt_queue_s[10];*/

    int ff = threadIdx.x + blockIdx.x*blockDim.x;// + offset;
    int ii = threadIdx.y + blockIdx.y*blockDim.y;
/*
    if(ff < size_eff && ff+offset < wgt_queue_size){
        s_wgt_queue[ff] = dev.wgt_queue[ff+offset];
        s_wgt_queue_k[ff] = dev.wgt_queue_k[ff+offset];
        s_wgt_queue_r[ff] = dev.wgt_queue_r[ff+offset];
        s_wgt_queue_s[ff] = dev.wgt_queue_s[ff+offset];
    }   
    __syncthreads();
*/
    if(ii < *act_queue_size && ff < size_eff){
        float act = dev.act_queue[ii];
        int x = dev.act_queue_x[ii];
        int y = dev.act_queue_y[ii];

        float wgt = d_wgt_queue[ff];//dev.wgt_queue[ff];
        int k = d_wgt_queue_k[ff];//dev.wgt_queue_k[ff];
        int r = d_wgt_queue_r[ff];//dev.wgt_queue_r[ff];
        int s = d_wgt_queue_s[ff];//dev.wgt_queue_s[ff];

        //works for power of 2 strides
        int w = (x-r) >> log2(stride);
        int h = (y-s) >> log2(stride);

        if(w >= 0 && w < W && h >= 0 && h < H) {
            int pos = n * W * H * K + k * W * H + w * H + h;
            //TODO: memory access not coalesced
            //TODO: try to remove atomicAdd
            //d_output_activations[pos] += act * wgt;
            atomicAdd(d_output_activations + pos, act * wgt);
        }
    }
}

//############################################### CPU SCNN #############################################################

void addBias(int N, int K, int W, int H, const Layer &layer, float *d_output_activations) {

    #ifndef GLOBAL_TIME
    double timeStampA = getTimeStamp();
    #endif

    float *d_bias = host2Dev(layer.getMaxIndex("bias"), layer.bias,"allocate device bias");

    dim3 block(16, 16, 4);
    dim3 grid((H+block.x-1)/block.x,(W+block.y-1)/block.y,(K+block.z-1)/block.z);
    check_grid(grid,"addBias");

    hipStream_t streams[N+1];

    for(int n=0; n< N; n++){
        hipStreamCreate(&streams[n+1]);
        kAddBias<<<grid, block,0,streams[n+1]>>>(n,K,W,H,d_bias,d_output_activations);
    }
    hipDeviceSynchronize();

    check_error(hipFree(d_bias),"free device bias");

    #ifndef GLOBAL_TIME
    double timeStampB = getTimeStamp();
    printf("kAddBias block: (%d,%d,1), grid: (%d,%d,1)\n",block.x,block.y,grid.x,grid.y);
    printf("kAddBias time %.6f\n",(timeStampB-timeStampA));
    #endif
}

void relu(int N, int K, int W, int H, const Layer &layer, float *d_output_activations) {

    #ifndef GLOBAL_TIME
    double timeStampA = getTimeStamp();
    #endif

    dim3 block(1024, 1);
    dim3 grid((K*W*H+block.x-1)/block.x,1);
    
    if(layer.ReLU){
        check_grid(grid,"relu");
        hipStream_t streams[N+1];
        for(int n = 0; n < N; n++){
            hipStreamCreate(&streams[n+1]);
            kRelu<<<grid,block,0,streams[n+1]>>>(N,K,W,H,d_output_activations);
        }
        hipDeviceSynchronize();
    }

    #ifndef GLOBAL_TIME
    double timeStampB = getTimeStamp();
    printf("kRelu block: (%d,%d,1), grid: (%d,%d,1)\n",block.x,block.y,grid.x,grid.y);
    printf("kRelu time %.6f\n",(timeStampB-timeStampA));
    #endif
}

void populate_effectual_activations(int n, int channel, int sx, int sy, int stride, const Layer &layer, 
		device_data dev, int *act_queue_size) {

    #ifndef GLOBAL_TIME
    double timeStampA = getTimeStamp();
    #endif

    int C = (int) layer.act_shape[1];
    int X = (int) layer.act_shape[2];
    int Y = (int) layer.act_shape[3];

    dim3 block(32, 32);
    dim3 grid((Y+block.x-1)/block.x,(X+block.y-1)/block.y);
    check_grid(grid,"populate_effectual_activations");

    //TODO:add streams
    kPopulate_effectual_activations<<<grid,block>>>(n,channel,sx,sy,C,X,Y,stride,dev,act_queue_size);
    hipDeviceSynchronize();

    #ifndef GLOBAL_TIME
    double timeStampB = getTimeStamp();
    printf("kPopulate_effectual_activations block: (%d,%d,1), grid: (%d,%d,1)\n",block.x,block.y,grid.x,grid.y);
    printf("kPopulate_effectual_activations time %.6f\n",(timeStampB-timeStampA));
    #endif
}

void computePE(int n, int W, int H, int K, int stride, int act_queue_size, int wgt_queue_size, int *d_act_queue_size, 
		device_data dev, int size_eff, int offset, float *d_output_activations, hipStream_t stream) {

    #ifndef GLOBAL_TIME
    double timeStampA = getTimeStamp();
    #endif

    //block size might be different for conv and fc
    dim3 block(128, 8);
    dim3 grid((size_eff+block.x-1)/block.x,(act_queue_size+block.y-1)/block.y);
    check_grid(grid,"computePE");

    kComputePE<<<grid,block,0,stream>>>(n,W,H,K,stride,d_act_queue_size,wgt_queue_size,dev.wgt_queue+offset,dev.wgt_queue_k+offset,
        dev.wgt_queue_r+offset,dev.wgt_queue_s+offset,size_eff,dev,d_output_activations);
    //hipDeviceSynchronize();

    #ifndef GLOBAL_TIME
    double timeStampB = getTimeStamp();
    printf("kComputePE block: (%d,%d,1), grid: (%d,%d,1)\n",block.x,block.y,grid.x,grid.y);
    printf("kComputePE time %.6f\n",(timeStampB-timeStampA));
    #endif

}

void computeTile(int n, int ct, int ck, int kc, int Kc, int X, int Y, int K, int W, int H, int R, int S,
        const Layer &layer, const host_data &hst, device_data dev, float *d_output_activations) {

    int stride = layer.stride;

    // Iterate strides
    for(int sx = 0; sx < stride; sx++) {
        for(int sy = 0; sy < stride; sy++) {

			// Transfer working weights to GPU
        	int pos = (ct+ck)*stride*stride + sx*stride + sy;
        	/*check_error(hipMemcpy(dev.wgt_queue, hst.wgt_queue[pos], hst.wgt_queue_size[pos]*sizeof(float), hipMemcpyHostToDevice),
           		"copy weights queue from host to device");
        	check_error(hipMemcpy(dev.wgt_queue_k, hst.wgt_queue_k[pos], hst.wgt_queue_size[pos]*sizeof(int), hipMemcpyHostToDevice),
           		"copy weights queue from host to device");
			check_error(hipMemcpy(dev.wgt_queue_r, hst.wgt_queue_r[pos], hst.wgt_queue_size[pos]*sizeof(int), hipMemcpyHostToDevice),
           		"copy weights queue from host to device");
        	check_error(hipMemcpy(dev.wgt_queue_s, hst.wgt_queue_s[pos], hst.wgt_queue_size[pos]*sizeof(int), hipMemcpyHostToDevice),
           		"copy weights queue from host to device");*/

            int *d_act_queue_size;
    		check_error(hipMalloc((void**) &d_act_queue_size, sizeof(int)),"allocate activations queue size");
    		check_error(hipMemset(d_act_queue_size,0, sizeof(int)),"set activations queue size to zero");

            // Populate activations queue
            populate_effectual_activations(n,ct+ck,sx,sy,stride,layer,dev,d_act_queue_size);

            //TODO optimize size usage (computePE needs to read it from mem, and we need to read it from host
            // in order to assign the block size
            int act_queue_size;
            check_error(hipMemcpy(&act_queue_size, d_act_queue_size, sizeof(int), hipMemcpyDeviceToHost),
                "copy activation queue size from device to host");
            
            int streamSize = 20000;
            int nStreams = (hst.wgt_queue_size[pos]+streamSize-1)/streamSize;
            hipStream_t streams[nStreams+1];
            int offset = 0, size_eff = 0;

            for(int i = 0; i< nStreams;i++){
                offset = i*streamSize;
                size_eff = (offset+streamSize > hst.wgt_queue_size[pos])? hst.wgt_queue_size[pos]-offset : streamSize;
                hipStreamCreate(&streams[i+1]);

                check_error(hipMemcpyAsync(dev.wgt_queue+offset, hst.wgt_queue[pos]+offset, size_eff*sizeof(float), hipMemcpyHostToDevice, streams[i+1]),
                    "copy weights queue from host to device");
                check_error(hipMemcpyAsync(dev.wgt_queue_k+offset, hst.wgt_queue_k[pos]+offset, size_eff*sizeof(int), hipMemcpyHostToDevice, streams[i+1]),
                    "copy weights queue from host to device");
                check_error(hipMemcpyAsync(dev.wgt_queue_r+offset, hst.wgt_queue_r[pos]+offset, size_eff*sizeof(int), hipMemcpyHostToDevice, streams[i+1]),
                    "copy weights queue from host to device");
                check_error(hipMemcpyAsync(dev.wgt_queue_s+offset, hst.wgt_queue_s[pos]+offset, size_eff*sizeof(int), hipMemcpyHostToDevice, streams[i+1]),
                    "copy weights queue from host to device");
                //do actual convolution
                //can be done better (remove dev and pass the wgt tiles only ?)
                //computePE(n,W,H,K,stride,act_queue_size,hst.wgt_queue_size[pos],d_act_queue_size,dev,size_eff,offset,d_output_activations,streams[i+1]);
                computePE(n,W,H,K,stride,act_queue_size,hst.wgt_queue_size[pos],d_act_queue_size,dev,size_eff,offset,d_output_activations,streams[i+1]);
            }

            //hipDeviceSynchronize();

            //do actual convolution
            //computePE(n,W,H,K,stride,act_queue_size,hst.wgt_queue_size[pos],d_act_queue_size,dev,d_output_activations);

            //free GPU resources
            check_error(hipFree(d_act_queue_size),"free device activations size");

        }
    }
}

//############################################### Main #################################################################

int main(int argc, char *argv[]) {

    double total_time = 0.0;

    std::vector<Layer> network = read_bvlc_alexnet();
    //std::vector<Layer> network = read_vgg_cnn_s();

    for(int i = 0; i < network.size(); i++) {

    	Layer layer = network[i];
    
        layer.read_layer();

        if(layer.type == "fc") {
            layer.reshape_to_2D();
            int C = layer.act_shape[1];
            layer.act_split_4D((unsigned)(C / 256), 16, 16);

            int Ck = layer.wgt_shape[1];
            layer.wgt_split_4D((unsigned)(Ck / 256), 16, 16);
        }

        layer.zero_pad();
        #ifdef FORCE_ONE_IMAGE
        int N = 1;
        #else
        int N = (int) layer.act_shape[0];
        #endif
        int C = (int) layer.act_shape[1];
        int X = (int) layer.act_shape[2];
        int Y = (int) layer.act_shape[3];

        int K = (int) layer.wgt_shape[0];
        int Ck = (int) layer.wgt_shape[1];
        int R = (int) layer.wgt_shape[2];
        int S = (int) layer.wgt_shape[3];

		int padding = layer.padding;
        int stride = layer.stride;

        int W = (X - R)/stride + 1;
        int H = (Y - S)/stride + 1;

        int groups = C / Ck;
        int Kc = K / groups;
        int kc = 0;

        // Allocate compressed weights off-line
		host_data hst;

        for(int ct = 0; ct < C; ct+=Ck) {
            for(int ck = 0; ck < Ck; ck++) {
            	for(int sx = 0; sx < stride; sx++) {
        			for(int sy = 0; sy < stride; sy++) {   

        				int wgt_queue_max_size = R*S*Kc;

        				int k_begin = kc;
    					int k_end = k_begin + Kc;

        			    int wgt_queue_size_ch = 0;
    					float *wgt_queue_ch;
						int *wgt_queue_k_ch, *wgt_queue_r_ch, *wgt_queue_s_ch;

    					hipHostMalloc((void **) &wgt_queue_ch, wgt_queue_max_size * sizeof(float));
			            if (wgt_queue_ch == NULL) {
			                fprintf(stderr, "Error: Failed to allocate weights queue!\n");
			                exit(EXIT_FAILURE);
			            }
    					hipHostMalloc((void **) &wgt_queue_k_ch, wgt_queue_max_size * sizeof(int));
			            if (wgt_queue_k_ch == NULL) {
			                fprintf(stderr, "Error: Failed to allocate weights queue k!\n");
			                exit(EXIT_FAILURE);
			            }
    					hipHostMalloc((void **) &wgt_queue_r_ch, wgt_queue_max_size * sizeof(int));
			            if (wgt_queue_r_ch == NULL) {
			                fprintf(stderr, "Error: Failed to allocate weights queue r!\n");
			                exit(EXIT_FAILURE);
			            }
    					hipHostMalloc((void **) &wgt_queue_s_ch, wgt_queue_max_size * sizeof(int));
			            if (wgt_queue_s_ch == NULL) {
			                fprintf(stderr, "Error: Failed to allocate weights queue s!\n");
			                exit(EXIT_FAILURE);
			            }

			            for(int r = 0; r < R; r++) {
			                int tmp_sx = (r + padding) % stride;
			                for(int s = 0; s < S; s++) {
			                    int tmp_sy = (s + padding) % stride;
			                    for(int k = k_begin; k < k_end; k++) {
			                        float wgt_bits = layer.wgt_get(k,ck,r,s);
			                        if (wgt_bits != 0 && sx == tmp_sx && sy == tmp_sy) {
			                            wgt_queue_ch[wgt_queue_size_ch] = wgt_bits;
			                            wgt_queue_k_ch[wgt_queue_size_ch] = k;
			                            wgt_queue_r_ch[wgt_queue_size_ch] = r;
			                            wgt_queue_s_ch[wgt_queue_size_ch] = s;
			                            wgt_queue_size_ch++;
			                        }
			                    }
			                }
			            }

			            hst.wgt_queue.push_back(wgt_queue_ch);
			            hst.wgt_queue_k.push_back(wgt_queue_k_ch);
			            hst.wgt_queue_r.push_back(wgt_queue_r_ch);
			            hst.wgt_queue_s.push_back(wgt_queue_s_ch);
			            hst.wgt_queue_size.push_back(wgt_queue_size_ch);

        			}
    			}
            }
            kc += Kc;
        }

        uint32_t bytes = N*K*W*H * sizeof(float);

        float *d_output_activations;
        check_error(hipMalloc((void **) &d_output_activations, bytes),"allocate device output activations");

        float *h_output_activations;
        check_error(hipHostMalloc((void **) &h_output_activations, bytes),"allocate output activations");

    	double timeStampA = getTimeStamp();

        addBias(N, K, W, H, layer, d_output_activations);

        ////////core compute/////////////
        // Allocate space for the queues on device (allocate once and reuse)
        float *d_act_queue, *d_wgt_queue;
        int *d_act_queue_x, *d_act_queue_y;
        int *d_wgt_queue_k, *d_wgt_queue_r, *d_wgt_queue_s;

    	float *d_act = host2Dev(layer.getMaxIndex("activations"), layer.activations,"copy device activations");

        //max. size is one activation channel
        check_error(hipMalloc((void**) &d_act_queue, X*Y*sizeof(float)),"allocate device activations queue");
        check_error(hipMalloc((void**) &d_act_queue_x, X*Y*sizeof(int)),"allocate device activations queue X dim");
        check_error(hipMalloc((void**) &d_act_queue_y, X*Y*sizeof(int)),"allocate device activations queue Y dim");

        //max. size is the numebr of kernel channels processed in parallel with each activation channel
        check_error(hipMalloc((void**) &d_wgt_queue, Kc*R*S*sizeof(float)),"allocate device weights queue");
        check_error(hipMalloc((void**) &d_wgt_queue_k, Kc*R*S*sizeof(int)),"allocate device weights queue K filter");
        check_error(hipMalloc((void**) &d_wgt_queue_r, Kc*R*S*sizeof(int)),"allocate device weights queue R dim");
        check_error(hipMalloc((void**) &d_wgt_queue_s, Kc*R*S*sizeof(int)),"allocate device weights queue S dim");

		//copy to struct
		device_data dev;
		dev.act = d_act;
		dev.act_queue = d_act_queue;
		dev.act_queue_x = d_act_queue_x;
		dev.act_queue_y = d_act_queue_y;
		dev.wgt_queue = d_wgt_queue;
		dev.wgt_queue_k = d_wgt_queue_k;
		dev.wgt_queue_r = d_wgt_queue_r;
		dev.wgt_queue_s = d_wgt_queue_s;		

        for(int n = 0; n < N; n++) {
            kc = n;
            for(int ct = 0; ct < C; ct+=Ck) {
                for(int ck = 0; ck < Ck; ck++) {
                    computeTile(n,ct,ck,kc,Kc,X,Y,K,W,H,R,S,layer,hst,dev,d_output_activations);
                }
                kc += Kc;
            }
        }

        relu(N, K, W, H, layer, d_output_activations);

        check_error(hipMemcpy(h_output_activations, d_output_activations, bytes, hipMemcpyDeviceToHost),
                "copy output activations from device to host");

		double timeStampB = getTimeStamp();
		printf("Layer %s time: %.6f\n",layer.name.c_str(),timeStampB-timeStampA);
		total_time += timeStampB-timeStampA;

        //free GPU resources
        check_error(hipFree(d_act),"free device activations");
        
        check_error(hipFree(d_act_queue),"free device activations queue");
        check_error(hipFree(d_act_queue_x),"free device activations queue X dim");
        check_error(hipFree(d_act_queue_y),"free device activations queue Y dim");
        
        check_error(hipFree(d_wgt_queue),"free device weights queue");
        check_error(hipFree(d_wgt_queue_k),"free device weights queue K dim");
        check_error(hipFree(d_wgt_queue_r),"free device weights queue R dim");
        check_error(hipFree(d_wgt_queue_s),"free device weights queue S dim");
        ///////////////////////////

		for(int ck = 0; ck < C; ck++) {
           	for(int sx = 0; sx < stride; sx++) {
        		for(int sy = 0; sy < stride; sy++) {  

       				int pos = ck*stride*stride + sx*stride + sy;

                    hipHostFree(hst.wgt_queue[pos]);
		            hipHostFree(hst.wgt_queue_k[pos]);
		            hipHostFree(hst.wgt_queue_r[pos]);
		            hipHostFree(hst.wgt_queue_s[pos]);
        		}
        	}
        }

        check_values(layer,h_output_activations);
        hipHostFree(h_output_activations);

    }

	printf("Total time: %.6f\n",total_time);

    return 0;
}
