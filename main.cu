#include "tensor.h"
#include "layer.h"
#include <cmath>

std::vector<Layer> read_bvlc_alexnet() {
    std::vector<Layer> network;
    network.emplace_back(Layer("bvlc_alexnet","conv1","conv",true,4,0));
    network.emplace_back(Layer("bvlc_alexnet","conv2","conv",true,1,2));
    network.emplace_back(Layer("bvlc_alexnet","conv3","conv",true,1,1));
    network.emplace_back(Layer("bvlc_alexnet","conv4","conv",true,1,1));
    network.emplace_back(Layer("bvlc_alexnet","conv5","conv",true,1,1));
    network.emplace_back(Layer("bvlc_alexnet","fc6","fc",true,1,0));
    network.emplace_back(Layer("bvlc_alexnet","fc7","fc",true,1,0));
    network.emplace_back(Layer("bvlc_alexnet","fc8","fc",false,1,0));
    return network;
};

// MAIN

int main(int argc, char *argv[]) {

    auto network = read_bvlc_alexnet();

    for(auto layer : network) {

        read_layer(layer);

        }

return 0;
}